#include "hip/hip_runtime.h"
#include <Eigen/Dense>

#if DECOMPOSITION==USE_SPARSE_QR
#include <Eigen/SparseQR>
#include <Eigen/Sparse>
#endif

// #include <vector>

#include "../interface/fnnls.h"
#include "../interface/vector.h"

#ifdef DEBUG_FNNLS_GPU
#include <stdio.h>
#endif

using namespace Eigen;


__host__ __device__
void print_fixed_matrix(const FixedMatrix &M) {
    printf("ciao");
    for(unsigned int i = 0; i < MATRIX_SIZE; i++){
        for(unsigned int j = 0; j < MATRIX_SIZE; j++){                   
            printf("%d ", M(i,j));
        }
        printf("\n");
    }
}

__host__ __device__
void print_fixed_vector(const FixedVector &V) {
    printf("ciao");
    for(unsigned int i = 0; i < MATRIX_SIZE; i++){
        printf("%d ", V[i]);
    }
        printf("\n");
}


__device__ __host__ FixedVector fnnls(const FixedMatrix &A, const FixedVector &b, const double eps, const unsigned int max_iterations){

	#ifdef DEBUG_FNNLS_GPU
	printf("debug fnnls");
	print_fixed_matrix(A);
	print_fixed_vector(b);
	#else
    printf("hello world\n");
    #endif
	
 	// Fast NNLS (fnnls) algorithm as per 
	// http://users.wfu.edu/plemmons/papers/Chennnonneg.pdf
	// page 8
	
	// FNNLS memorizes the A^T * A and A^T * b to reduce the computation.
	// The pseudo-inverse obtined has the same numerical problems so
	// I keep the same decomposition utilized for NNLS.

	
	// pseudoinverse (A^T * A)^-1 * A^T 
	// this pseudo-inverse has numerical issues
	// in order to avoid that I substitued the pseudoinvese wiht the QR decomposition
	
	#if DECOMPOSITION==USE_SPARSE_QR
	Eigen::SparseQR<Eigen::SparseMatrix<double>, Eigen::VectorXd> solver;
	#elif DECOMPOSITION==USE_LLT
	Eigen::LLT<FixedMatrix> solver;
	#elif DECOMPOSITION==USE_HOUSEHOLDER
	Eigen::HouseholderQR<FixedMatrix> solver;
	#endif
	
	vector<unsigned int> P;
	vector<unsigned int> R(VECTOR_SIZE);

	// initial set of indexes
	#pragma unroll
	for ( unsigned int i=0; i<VECTOR_SIZE; ++i) R[i] = i;

	// initial solution vector
	FixedVector x = FixedVector::Zero();

	auto AtA = A.transpose() * A;
	auto Atb = A.transpose() * b;

	// main loop 
	for (int iter=0; iter<max_iterations; ++iter){

	
		// FNNLS
		FixedVector w = Atb - (AtA*x);
		

		// initialize the value for the while guard
		// max_index will contain the index of the max coeff anf max_w is the max coeff 
		unsigned int max_index = R[0];	
		unsigned int remove_index = 0;

		for (unsigned int i=0; i<R.size(); ++i){
			auto index = R[i];
			if(w[index] > w[max_index]){
				max_index = index;
				remove_index = i;
			}
		}


		P.push_back(max_index);
		// R.erase(R.begin()+remove_index);
		R.erase(remove_index);

		// termination condition
		if(R.empty() || w[max_index] < eps) break;


		FixedMatrix A_P = FixedMatrix::Zero();

		for(auto index: P) A_P.col(index)=A.col(index);

		#if DECOMPOSITION==USE_SPARSE_QR
		solver.compute(A_P.sparseView());
		#else
		solver.compute(A_P);
		#endif

		Eigen::VectorXd s =  solver.solve(b);
		
		// inner loop
		while(true){

			auto min_s = std::numeric_limits<double>::max();

			for (auto index: P)
				min_s = std::min(s[index],min_s);
			

			if(min_s > 0 ) break;

			auto alpha = std::numeric_limits<double>::max();

			for (auto index: P){
				if (s[index] <= 0 ){
					alpha = -std::min(x[index]/(x[index]-s[index]), alpha);
				}
			}

			for (auto index: P)
				x[index] += alpha*(s[index]-x[index]);

			vector<unsigned int> tmp;

			for(int i=P.size()-1; i>=0; --i){
				auto index = P[i]; 
				if(x[index]==0){
					R.push_back(index);
					tmp.push_back(i);
				}
			}

			// for(auto index: tmp) P.erase(P.begin()+index);
			for(auto index: tmp) P.erase(index);
			
			A_P.setZero();
	
			for(auto index: P) A_P.col(index)=A.col(index);
			
			#if DECOMPOSITION==USE_SPARSE_QR
			solver.compute(A_P.sparseView());
			#else
			solver.compute(A_P);
			#endif
			
			s =  solver.solve(b);

			for(auto index: R) s[index]=0;

			return x;
		}

		x = s;
	}

	#ifdef DEBUG_FNNLS_GPU
	print_fixed_vector(x);
	#endif

	return x;
}

__global__ void fnnls_kernel(NNLS_args *args, FixedVector* x, unsigned int n, double eps, unsigned int max_iterations){
	// thread idx
    printf("hello fnnls\n");
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	printf("thread index %i\n", i);
	if (i>=n) return;
	auto &A = args[i].A;
	auto &b = args[i].b;
	x[i] = fnnls(A, b, eps, max_iterations);

}
