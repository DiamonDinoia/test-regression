#include "hip/hip_runtime.h"
#include "../interface/fnnls.h"

#include <Eigen/Dense>
#include <Eigen/SparseQR>
#include <Eigen/Sparse>

#include <iostream>

#ifdef DEBUG
#include <vector>
#endif

#include "../interface/nnls.h"


using namespace std;
using namespace Eigen;

__device__ __host__ FixedVector fnnls(const FixedMatrix &A, const FixedVector &b, const double eps, const unsigned int max_iterations){

 	// Fast NNLS (fnnls) algorithm as per 
	// http://users.wfu.edu/plemmons/papers/Chennnonneg.pdf
	// page 8
	
	// FNNLS memorizes the A^T * A and A^T * b to reduce the computation.
	// The pseudo-inverse obtined has the same numerical problems so
	// I keep the same decomposition utilized for NNLS.

	
	// pseudoinverse (A^T * A)^-1 * A^T 
	// this pseudo-inverse has numerical issues
	// in order to avoid that I substitued the pseudoinvese wiht the QR decomposition
	
	Eigen::LLT<FixedMatrix> solver;
	// Eigen::SparseQR<Eigen::SparseMatrix<double>, Eigen::VectorXd> solver;
	
	std::vector<unsigned int> P;
	std::vector<unsigned int> R(VECTOR_SIZE);

	// initial set of indexes
	#pragma unroll
	for ( unsigned int i=0; i<VECTOR_SIZE; ++i) R[i] = i;

	// initial solution vector
	FixedVector x = FixedVector::Zero();

	auto AtA = A.transpose() * A;
	auto Atb = A.transpose() * b;

	// main loop 
	for (int iter=0; iter<max_iterations; ++iter){

		#ifdef DEBUG
		// cout << "iter " << iter << endl;
		#endif
	
		// FNNLS
		FixedVector w = Atb - (AtA*x);
		
		#ifdef DEBUG
		// cout << "w" << endl << w << endl;
		#endif

		// initialize the value for the while guard
		// max_index will contain the index of the max coeff anf max_w is the max coeff 
		unsigned int max_index = R[0];	
		unsigned int remove_index = 0;

		for (unsigned int i=0; i<R.size(); ++i){
			auto index = R[i];
			if(w[index] > w[max_index]){
				max_index = index;
				remove_index = i;
			}
		}

		#ifdef DEBUG
		// cout << "max index " << max_index << endl;
		#endif

		P.emplace_back(max_index);
		R.erase(R.begin()+remove_index);

		// termination condition
		if(R.empty() || w[max_index] < eps) break;

		#ifdef DEBUG
		// cout << "P " << endl;
		// for (auto elem : P) cout << elem << " ";
		// cout << endl;
		// cout << "R " << endl;
		// for (auto elem : R) cout << elem << " ";
		// cout << endl;
		#endif

		FixedMatrix A_P = FixedMatrix::Zero();

		for(auto index: P) A_P.col(index)=A.col(index);

		// solver.compute(A_P.sparseView());
		solver.compute(A_P);

		#ifdef DEBUG
		// cout << "A_P " << endl << A_P << endl; 
		#endif

		Eigen::VectorXd s =  solver.solve(b);
		
		#ifdef DEBUG
		// cout << "s" << endl << s << endl;
		#endif

		// inner loop
		while(true){

			auto min_s = std::numeric_limits<double>::max();

			for (auto index: P)
				min_s = std::min(s[index],min_s);
			
			#ifdef DEBUG
			// cout << "min_s " << min_s << endl;
			#endif

			if(min_s > 0 ) break;

			auto alpha = std::numeric_limits<double>::max();

			for (auto index: P){
				if (s[index] <= 0 ){
					alpha = -std::min(x[index]/(x[index]-s[index]), alpha);
				}
			}
			#ifdef DEBUG

			// cout << "alpha " << alpha << endl;

			// cout << "x before" << endl << x << endl;

			#endif

			for (auto index: P)
				x[index] += alpha*(s[index]-x[index]);

			#ifdef DEBUG
			// cout << "x after" << endl << x << endl;
			#endif

			std::vector<unsigned int> tmp;

			#ifdef DEBUG
			// cout << "P  before" << endl;
			// for (auto elem : P) cout << elem << " ";
			// cout << endl;
			// cout << "R before" << endl;
			// for (auto elem : R) cout << elem << " ";
			// cout << endl;
			#endif


			for(int i=P.size()-1; i>=0; --i){
				auto index = P[i]; 
				if(x[index]==0){
					R.emplace_back(index);
					tmp.emplace_back(i);
				}
			}

			for(auto index: tmp) P.erase(P.begin()+index);
			
			#ifdef DEBUG

			// cout << "P  after" << endl;
			// for (auto elem : P) cout << elem << " ";
			// cout << endl;
			// cout << "R after" << endl;
			// for (auto elem : R) cout << elem << " ";
			// cout << endl;
			#endif

			A_P.setZero();
	
			for(auto index: P) A_P.col(index)=A.col(index);
			
			solver.compute(A_P.sparseView());

			s =  solver.solve(b);

			for(auto index: R) s[index]=0;

			return x;
		}

		x = s;
	}


	return x;
}

