#include "hip/hip_runtime.h"
#include "../interface/kernels.h"

__global__ void nnls_kernel(NNLS_args *args, FixedVector* x, unsigned int n, double eps, unsigned int max_iterations){
        // thread idx
        int i = blockIdx.x*blockDim.x + threadIdx.x;
        if (i>=n) return;
        auto &A = args[i].A;
        auto &b = args[i].b;
        x[i] = nnls(A, b, eps, max_iterations);
    }

__global__ void fnnls_kernel(NNLS_args *args, FixedVector* x, unsigned int n, double eps, unsigned int max_iterations){
        // thread idx
        int i = blockIdx.x*blockDim.x + threadIdx.x;
        if (i>=n) return;
        auto &A = args[i].A;
        auto &b = args[i].b;
        x[i] = fnnls(A, b, eps, max_iterations);
    }