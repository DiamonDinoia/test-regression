#include "hip/hip_runtime.h"
#include <Eigen/Dense>
#include <Eigen/SparseQR>
#include <Eigen/Sparse>

#include <vector>

#ifdef DEBUG_FNNLS
#include <iostream>
#endif

#include "../interface/nnls.h"

using namespace std;
using namespace Eigen;


__device__ __host__ FixedVector nnls(const FixedMatrix &A, const FixedVector &b, const double eps, const unsigned int max_iterations){

 	// Fast NNLS (fnnls) algorithm as per 
	// http://users.wfu.edu/plemmons/papers/Chennnonneg.pdf
	// page 8
	

	// pseudoinverse (A^T * A)^-1 * A^T 
	// this pseudo-inverse has numerical issues
	// in order to avoid that I substitued the pseudoinvese wiht the QR decomposition
	
	// Eigen::SparseQR<Eigen::SparseMatrix<double>, Eigen::VectorXd> solver;
	Eigen::LLT<FixedMatrix> solver;
	
	thrust::device_vector<unsigned int> P;
	thrust::device_vector<unsigned int> R(VECTOR_SIZE);

	// initial set of indexes
	#pragma unroll
	for ( unsigned int i=0; i<VECTOR_SIZE; ++i) R[i] = i;

	// initial solution vector
	FixedVector x = FixedVector::Zero();


	// main loop 
	for (int iter=0; iter<max_iterations; ++iter){

		#ifdef DEBUG_FNNLS
		// cout << "iter " << iter << endl;
		#endif

		//NNLS
		// initialize the cost vector
		FixedVector w = A.transpose()*(b - (A*x));
		
		#ifdef DEBUG_FNNLS
		// cout << "w" << endl << w << endl;
		#endif

		// initialize the value for the while guard
		// max_index will contain the index of the max coeff anf max_w is the max coeff 
		unsigned int max_index = R[0];	
		unsigned int remove_index = 0;

		for (unsigned int i=0; i<R.size(); ++i){
			auto index = R[i];
			if(w[index] > w[max_index]){
				max_index = index;
				remove_index = i;
			}
		}

		#ifdef DEBUG_FNNLS
		// cout << "max index " << max_index << endl;
		#endif

		P.push_back(max_index);
		R.erase(R.begin()+remove_index);

		// termination condition
		if(R.empty() || w[max_index] < eps) break;

		#ifdef DEBUG_FNNLS
		// cout << "P " << endl;
		// for (auto elem : P) cout << elem << " ";
		// cout << endl;
		// cout << "R " << endl;
		// for (auto elem : R) cout << elem << " ";
		// cout << endl;
		#endif

		FixedMatrix A_P = FixedMatrix::Zero();


		for(auto index: P) A_P.col(index)=A.col(index);

		solver.compute(A_P);

		#ifdef DEBUG_FNNLS
		// cout << "A_P " << endl << A_P << endl; 
		#endif

		// FixedVector s = (A_P.transpose()*A_P).inverse() * A_P.transpose() * b;
		Eigen::VectorXd s =  solver.solve(b);
	
		for(auto index: R) s[index]=0;

		#ifdef DEBUG_FNNLS
		// cout << "s" << endl << s << endl;
		#endif

		// inner loop
		while(true){

			auto min_s = std::numeric_limits<double>::max();

			for (auto index: P)
				min_s = std::min(s[index],min_s);
			
			#ifdef DEBUG_FNNLS
			cout << "min_s " << min_s << endl;
			#endif

			if(min_s > 0 ) break;

			auto alpha = std::numeric_limits<double>::max();

			for (auto index: P){
				if (s[index] <= 0 ){
					alpha = -std::min(x[index]/(x[index]-s[index]), alpha);
				}
			}
			#ifdef DEBUG_FNNLS

			cout << "alpha " << alpha << endl;

			cout << "x before" << endl << x << endl;

			#endif

			for (auto index: P)
				x[index] += alpha*(s[index]-x[index]);

			#ifdef DEBUG_FNNLS
			cout << "x after" << endl << x << endl;
			#endif

			thrust::device_vector<unsigned int> tmp;

			#ifdef DEBUG_FNNLS
			// cout << "P  before" << endl;
			// for (auto elem : P) cout << elem << " ";
			// cout << endl;
			// cout << "R before" << endl;
			// for (auto elem : R) cout << elem << " ";
			// cout << endl;
			#endif


			for(int i=P.size()-1; i>=0; --i){
				auto index = P[i]; 
				if(x[index]==0){
					R.push_back(index);
					tmp.push_back(i);
				}
			}

			for(auto index: tmp) P.erase(P.begin()+index);
			
			#ifdef DEBUG_FNNLS

			// cout << "P  after" << endl;
			// for (auto elem : P) cout << elem << " ";
			// cout << endl;
			// cout << "R after" << endl;
			// for (auto elem : R) cout << elem << " ";
			// cout << endl;
			#endif

			// NNLS

			A_P.setZero();
	
			for(auto index: P) A_P.col(index)=A.col(index);
			
			solver.compute(A_P.sparseView());

			s =  solver.solve(b);

			for(auto index: R) s[index]=0;

			return x;
		}

		x = s;
	}


	return x;
}
