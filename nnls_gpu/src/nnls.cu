#include "hip/hip_runtime.h"
#include <Eigen/Dense>

#ifdef DEBUG_NNLS_GPU
#include <stdio.h>
#endif

#include "../interface/nnls.h"
#include "../interface/vector.h"

using namespace Eigen;


__device__ __host__ 
void print_fixed_matrix(const FixedMatrix& M) {
  for (unsigned int i = 0; i < MATRIX_SIZE; i++) {
    for (unsigned int j = 0; j < MATRIX_SIZE; j++) {
      printf("%d ", M(i, j));
    }
    printf("\n");
  }
}

__device__ __host__ 
void print_fixed_vector(const FixedVector& V) {
  for (unsigned int i = 0; i < MATRIX_SIZE; i++) {
    printf("%d ", V[i]);
  }
  printf("\n");
}

__device__ __host__ 
void print_device_vector(vector<unsigned int> v){
	for(auto elem: v) printf("%u ", elem);
	printf("\n");

}



__device__ __host__
FixedVector nnls(const FixedMatrix& A,
                 const FixedVector& b,
                 const double eps,
                 const unsigned int max_iterations) {

#ifdef DEBUG_NNLS_GPU
  printf("nnls launched\n");
	printf("parameters size: A(%i,%i) b(%i)\n", A.rows(), A.cols(), b.cols());
	printf("A\n");
	print_fixed_matrix(A);
	printf("b\n");
	print_fixed_vector(b);
#endif

  // Fast NNLS (fnnls) algorithm as per
  // http://users.wfu.edu/plemmons/papers/Chennnonneg.pdf
  // page 8

  // pseudoinverse (A^T * A)^-1 * A^T
  // this pseudo-inverse has numerical issues
  // in order to avoid that I substitued the pseudoinvese wiht the QR
  // decomposition

	vector<unsigned int> P;
	vector<unsigned int> R(VECTOR_SIZE);

// initial set of indexes
	#pragma unroll
  for (unsigned int i = 0; i < VECTOR_SIZE; ++i) R[i] = i;
	
  #ifdef DEBUG_NNLS_GPU
	printf("P ");
	print_device_vector(P);
	printf("R ");
	print_device_vector(R);
  #endif
  // initial solution vector
  FixedVector x = FixedVector::Zero();

  // main loop
  for (int iter = 0; iter < max_iterations; ++iter) {
    // NNLS
    // initialize the cost vector

    FixedVector w = A.transpose() * (b - (A * x));


    // initialize the value for the while guard
    // max_index will contain the index of the max coeff anf max_w is the max
    // coeff
    unsigned int max_index = R[0];
    unsigned int remove_index = 0;

    #ifdef DEBUG_NNLS_GPU
    printf("Max index %u, remove index %u\n", max_index, remove_index);
    #endif
    for (unsigned int i = 0; i < R.size(); ++i) {
			auto index = R[i];
      if (w(index) > w(max_index)) {
				max_index = index;
        remove_index = i;
      }
    }
    #ifdef DEBUG_NNLS_GPU
		printf("before the erase\n");
		printf("P ");
		print_device_vector(P);
		printf("R ");
		print_device_vector(R);
		
		printf("Max index %u, remove index %u\n", max_index, remove_index);
    #endif
		
    P.push_back(max_index);
    // R.erase(R.begin()+remove_index);
    R.erase(remove_index);

    #ifdef DEBUG_NNLS_GPU
		printf("after the erase\n");
		printf("P ");
		print_device_vector(P);
		printf("R ");
		print_device_vector(R);
		#endif
		// fflush(NULL);
    // termination condition
    if (R.empty() || w[max_index] < eps)
      break;

    FixedMatrix A_P = FixedMatrix::Zero();

    for (auto index : P)
      A_P.col(index) = A.col(index);

// FixedVector s = (A_P.transpose()*A_P).inverse() * A_P.transpose() * b;
#if DECOMPOSITION == USE_LLT
    FixedVector s = A_P.llt().matrixL().solve(b);
#elif DECOMPOSITION == USE_LDLT
    FixedVector s = A_P.ldlt().matrixL().solve(b);
#elif DECOMPOSITION == USE_HOUSEHOLDER
    FixedVector s = A_P.colPivHouseholderQr().solve(b);
#endif

    #ifdef DEBUG_NNLS_GPU
    printf("after the decomposition\n");
    #endif
    
    for (auto index : R)
    s[index] = 0;
    
    // inner loop
    while (true) {
      auto min_s = std::numeric_limits<double>::max();
      
      for (auto index : P)
      min_s = std::min(s[index], min_s);
      
			#ifdef DEBUG_FNNLS
      cout << "min_s " << min_s << endl;
			#endif
      
      if (min_s > 0)
      break;
      
      auto alpha = std::numeric_limits<double>::max();
      
      for (auto index : P) {
        if (s[index] <= 0) {
          alpha = std::min(-x[index] / (s[index] - x[index]), alpha);
        }
      }
      
      printf("alpha %d", alpha);
      // fflush(NULL);
      
      for (auto index : P)
        x[index] += alpha * (s[index] - x[index]);

      vector<unsigned int> tmp;

      for (int i = P.size() - 1; i >= 0; --i) {
        auto index = P[i];
        if (x[index] == 0) {
          R.push_back(index);
          tmp.push_back(i);
        }
      }

      // for(auto index: tmp) P.erase(P.begin()+index);
      for (auto index : tmp)
        P.erase(index);

      A_P.setZero();

      for (auto index : P)
        A_P.col(index) = A.col(index);

#if DECOMPOSITION == USE_LLT
      s = A_P.llt().matrixL().solve(b);
#elif DECOMPOSITION == USE_LDLT
      s = A_P.ldlt().matrixL().solve(b);
#elif DECOMPOSITION == USE_HOUSEHOLDER
      s = A_P.colPivHouseholderQr().solve(b);
#endif

      for (auto index : R)
        s[index] = 0;
    }

    x = s;
  }

  return x;
}

__global__ void nnls_kernel(NNLS_args* args,
                            FixedVector* x,
                            unsigned int n,
                            double eps,
                            unsigned int max_iterations) {
  // thread idx
  // printf("hello nnls\n");
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // printf("thread index %i\n", i);
  if (i >= n)
    return;
	auto& A = args[i].A;
	// printf("inside the kernel\n");
	// print_fixed_matrix(A);
	auto& b = args[i].b;
	// print_fixed_vector(b);
  x[i] = nnls(A, b, eps, max_iterations);
}
