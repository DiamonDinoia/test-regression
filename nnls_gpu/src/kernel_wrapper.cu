#include "../interface/kernel_wrapper.h"
#include "../interface/nnls.h"
#include "../interface/fnnls.h"

#include <iostream>
#include <string>

void assert_if_error(std::string const& name) {
    auto check = [&name](auto code) {
        if (code != hipSuccess) {
            std::cout << hipGetErrorString(code) << ' ';
            std::cout << "in " << name << std::endl;
            assert(false);
        }
    };

    check(hipGetLastError());
}

std::vector<FixedVector> nnls_wrapper(
                            std::vector<NNLS_args> const& args,
                            double eps,
                            unsigned int max_iterations){
        // host solution vector
        std::vector<FixedVector> x(args.size());
        
        // device pointers
        NNLS_args* d_args;
        FixedVector* d_x;
        
        // arguments allocation
        hipMalloc((void**) &d_args, sizeof(NNLS_args) * args.size());
        assert_if_error("nnls argument allocation");
        // results allocation
        hipMalloc((void**) &d_x, sizeof(FixedVector) * args.size());
        assert_if_error("nnls result allocation");
        
        
        // arguments copy
        hipMemcpy(d_args, args.data(), sizeof(NNLS_args) * args.size(), hipMemcpyHostToDevice);
        assert_if_error("nnls parameters copy");
        

        printf("launch kernel nnls\n");
        // nnls_kernel<<<args.size()+255/256, 256>>>(d_args, d_x, args.size(), eps, max_iterations);
        nnls_kernel<<<1,1>>>(d_args, d_x, args.size(), eps, max_iterations);
        hipDeviceSynchronize();
        assert_if_error("nnls kernel");
        printf("finish kernel nnls\n");
        
        // copy the results back from the device
        hipMemcpy(&(x[0]), d_x, sizeof(FixedVector) * args.size(), hipMemcpyDeviceToHost);
        
        // clear and exit
        hipFree(d_args);
        hipFree(d_x);

        return x;
    }
    
    std::vector<FixedVector> fnnls_wrapper(
                            std::vector<NNLS_args> const& args,
                            double eps,
                            unsigned int max_iterations){
        // host solution vector
        std::vector<FixedVector> x(args.size());
        
        // device pointers
        NNLS_args* d_args;
        FixedVector* d_x;
        
        // arguments allocation
        hipMalloc((void**) &d_args, sizeof(NNLS_args) * args.size());
        // results allocation
        hipMalloc((void**) &d_x, sizeof(FixedVector) * args.size());


        // arguments copy
        hipMemcpy(d_args, args.data(), sizeof(NNLS_args) * args.size(), hipMemcpyHostToDevice);
        
        printf("launch kernel fnnsl\n");
        // fnnls_kernel<<<args.size()+255/256, 256>>>(d_args, d_x, args.size(), eps, max_iterations);
        fnnls_kernel<<<1,1>>>(d_args, d_x, args.size(), eps, max_iterations);
        hipDeviceSynchronize();
        assert_if_error("fnnls");
        printf("finish kernel fnnls\n");
        
        // copy the results back from the device
        hipMemcpy(&(x[0]), d_x, sizeof(FixedVector) * args.size(), hipMemcpyDeviceToHost);

        // clear and exit
        hipFree(d_args);
        hipFree(d_x);

        return x;            

    }
