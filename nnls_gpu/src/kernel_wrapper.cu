#include "../interface/kernel_wrapper.h"
#include "../interface/nnls.h"
#include "../interface/fnnls.h"

#include <iostream>
#include <string>


using namespace std;

void assert_if_error(std::string const& name) {
    auto check = [&name](auto code) {
        if (code != hipSuccess) {
            std::cout << hipGetErrorString(code) << ' ';
            std::cout << "in " << name << std::endl;
            assert(false);
        }
    };

    check(hipGetLastError());
}

std::vector<FixedVector> nnls_wrapper(
                            std::vector<NNLS_args> const& args,
                            double eps,
                            unsigned int max_iterations){

        // hipDeviceSetLimit(hipLimitPrintfFifoSize, 0);
        // host solution vector
        std::vector<FixedVector> x(args.size());
        
        // device pointers
        NNLS_args* d_args;
        FixedVector* d_x;
        
        #ifdef DEBUG_NNLS_WRAPPER
        cout << "sizes: " << args.size() << endl;

        for(auto& arg: args){
            
            cout << "A" << endl;
            cout << arg.A << endl;
            cout << "b" << endl;
            cout << arg.b << endl;
        }
        #endif
        // arguments allocation
        if(hipMalloc(&d_args, sizeof(NNLS_args) * args.size()) != hipSuccess){
            cout << "first malloc failed" << endl;
            assert(false);
        }
        assert_if_error("nnls argument allocation");
        // results allocation
        if(hipMalloc(&d_x, sizeof(FixedVector) * args.size()) != hipSuccess){
            cout << "second malloc failed" << endl;
            assert(false);
        }
        assert_if_error("nnls result allocation");
        
        
        // arguments copy
        auto error = hipMemcpy(d_args, args.data(), sizeof(NNLS_args) * args.size(), hipMemcpyHostToDevice);
        if(error != hipSuccess){
            cout << "memcpy failed" << endl;
            assert(false);
        }

        assert_if_error("nnls parameters copy");
        

        printf("launch kernel nnls\n");

        int nthreadsPerBlock = 256;
        int nblocks = (args.size() + nthreadsPerBlock - 1) / nthreadsPerBlock;
        cout << "threads per block " << nthreadsPerBlock << 
        " blocks " << nblocks <<
        " input size " << args.size() << endl; 
        nnls_kernel<<<nblocks, nthreadsPerBlock>>>(d_args, d_x, args.size(), eps, max_iterations);
        // nnls_kernel<<<10,10>>>(d_args, d_x, args.size(), eps, max_iterations);
        // nnls_kernel<<<args.size(),1>>>(d_args, d_x, args.size(), eps, max_iterations);
        // nnls_kernel<<<10, 10>>>(d_args, d_x, args.size(), eps, max_iterations);
        hipDeviceSynchronize();
        assert_if_error("nnls kernel");
        printf("finish kernel nnls\n");
        
        // copy the results back from the device
        hipMemcpy(x.data(), d_x, sizeof(FixedVector) * args.size(), hipMemcpyDeviceToHost);
        
        // clear and exit
        hipFree(d_args);
        hipFree(d_x);
        #ifdef DEBUG_NNLS_WRAPPER
        for(const auto& result: x){
            cout << "x" << endl;
            cout << result.transpose() << endl;
            // break;
        }
        #endif
        return x;
    }
    
    std::vector<FixedVector> fnnls_wrapper(
                            std::vector<NNLS_args> const& args,
                            double eps,
                            unsigned int max_iterations){
        // host solution vector
        std::vector<FixedVector> x(args.size());
        
        // device pointers
        NNLS_args* d_args;
        FixedVector* d_x;
        
        // arguments allocation
        hipMalloc((void**) &d_args, sizeof(NNLS_args) * args.size());
        // results allocation
        hipMalloc((void**) &d_x, sizeof(FixedVector) * args.size());


        // arguments copy
        hipMemcpy(d_args, args.data(), sizeof(NNLS_args) * args.size(), hipMemcpyHostToDevice);
        
        printf("launch kernel fnnsl\n");
        int nthreadsPerBlock = 256;
        int nblocks = (args.size() + nthreadsPerBlock - 1) / nthreadsPerBlock;
        fnnls_kernel<<<nblocks, nthreadsPerBlock>>>(d_args, d_x, args.size(), eps, max_iterations);
        // fnnls_kernel<<<1,1>>>(d_args, d_x, args.size(), eps, max_iterations);
        hipDeviceSynchronize();
        assert_if_error("fnnls");
        printf("finish kernel fnnls\n");
        
        // copy the results back from the device
        hipMemcpy(x.data(), d_x, sizeof(FixedVector) * args.size(), hipMemcpyDeviceToHost);

        // clear and exit
        hipFree(d_args);
        hipFree(d_x);

        // for(const auto& result: x){
            // cout << "x" << endl;
            // cout << result.transpose() << endl;
            // break;
        // }
        
        return x;            

    }
