#include "../interface/kernel_wrapper.h"
#include "../interface/nnls.h"
#include "../interface/fnnls.h"



std::vector<FixedVector> nnls_wrapper(
                            std::vector<NNLS_args> const& args,
                            double eps,
                            unsigned int max_iterations){
        // host solution vector
        std::vector<FixedVector> x(args.size());
        
        // device pointers
        NNLS_args* d_args;
        FixedVector* d_x;
        
        // arguments allocation
        hipMalloc((void**) &d_args, sizeof(NNLS_args) * args.size());
        // results allocation
        hipMalloc((void**) &d_x, sizeof(FixedVector) * args.size());


        // arguments copy
        hipMemcpy(d_args, args.data(), sizeof(NNLS_args) * args.size(), hipMemcpyHostToDevice);
        
        
        nnls_kernel<<<1, 1>>>(d_args, d_x, args.size(), eps, max_iterations);
        
        // copy the results back from the device
        hipMemcpy(d_x, &(x[0]), sizeof(FixedVector) * args.size(), hipMemcpyDeviceToHost);
        
        // clear and exit
        hipFree(d_args);
        hipFree(d_x);

        return x;
    }
    
    std::vector<FixedVector> fnnls_wrapper(
                            std::vector<NNLS_args> const& args,
                            double eps,
                            unsigned int max_iterations){
        // host solution vector
        std::vector<FixedVector> x(args.size());
        
        // device pointers
        NNLS_args* d_args;
        FixedVector* d_x;
        
        // arguments allocation
        hipMalloc((void**) &d_args, sizeof(NNLS_args) * args.size());
        // results allocation
        hipMalloc((void**) &d_x, sizeof(FixedVector) * args.size());


        // arguments copy
        hipMemcpy(d_args, args.data(), sizeof(NNLS_args) * args.size(), hipMemcpyHostToDevice);
        
        
        fnnls_kernel<<<1, 1>>>(d_args, d_x, args.size(), eps, max_iterations);
        
        // copy the results back from the device
        hipMemcpy(d_x,  &(x[0]), sizeof(FixedVector) * args.size(), hipMemcpyDeviceToHost);

        // clear and exit
        hipFree(d_args);
        hipFree(d_x);

        return x;            

    }