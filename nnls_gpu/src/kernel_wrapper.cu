#include "../interface/kernel_wrapper.h"
#include "../interface/nnls.h"
#include "../interface/fnnls.h"

#include <iostream>
#include <string>

void assert_if_error(std::string const& name) {
    auto check = [&name](auto code) {
        if (code != hipSuccess) {
            std::cout << hipGetErrorString(code) << std::endl;
            std::cout << "in " << name << std::endl;
            assert(false);
        }
    };

    check(hipGetLastError());
}

std::vector<FixedVector> nnls_wrapper(
                            std::vector<NNLS_args> const& args,
                            double eps,
                            unsigned int max_iterations){
        // host solution vector
        std::vector<FixedVector> x(args.size());
        
        // device pointers
        NNLS_args* d_args;
        FixedVector* d_x;
        
        // arguments allocation
        hipMalloc((void**) &d_args, sizeof(NNLS_args) * args.size());
        // results allocation
        hipMalloc((void**) &d_x, sizeof(FixedVector) * args.size());


        // arguments copy
        hipMemcpy(d_args, args.data(), sizeof(NNLS_args) * args.size(), hipMemcpyHostToDevice);
        

        printf("launch kernel nnls\n");
        nnls_kernel<<<1, 1>>>(d_args, d_x, args.size(), eps, max_iterations);
        hipDeviceSynchronize();
        assert_if_error("nnls");
        printf("finish kernel nnls\n");
        
        // copy the results back from the device
        hipMemcpy(&(x[0]), d_x, sizeof(FixedVector) * args.size(), hipMemcpyDeviceToHost);
        
        // clear and exit
        hipFree(d_args);
        hipFree(d_x);

        return x;
    }
    
    std::vector<FixedVector> fnnls_wrapper(
                            std::vector<NNLS_args> const& args,
                            double eps,
                            unsigned int max_iterations){
        // host solution vector
        std::vector<FixedVector> x(args.size());
        
        // device pointers
        NNLS_args* d_args;
        FixedVector* d_x;
        
        // arguments allocation
        hipMalloc((void**) &d_args, sizeof(NNLS_args) * args.size());
        // results allocation
        hipMalloc((void**) &d_x, sizeof(FixedVector) * args.size());


        // arguments copy
        hipMemcpy(d_args, args.data(), sizeof(NNLS_args) * args.size(), hipMemcpyHostToDevice);
        
        printf("launch kernel fnnsl\n");
        fnnls_kernel<<<1, 1>>>(d_args, d_x, args.size(), eps, max_iterations);
        hipDeviceSynchronize();
        assert_if_error("fnnls");
        printf("finish kernel fnnls\n");
        
        // copy the results back from the device
        hipMemcpy(&(x[0]), d_x, sizeof(FixedVector) * args.size(), hipMemcpyDeviceToHost);

        // clear and exit
        hipFree(d_args);
        hipFree(d_x);

        return x;            

    }
