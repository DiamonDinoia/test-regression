#include "../interface/test_wrapper.h"
#include "../interface/test.h"


#include <iostream>
#include <string>
#include <cassert>

using namespace std;


void assert_if_error(std::string const& name) {
  auto check = [&name](auto code) {
      if (code != hipSuccess) {
          std::cout << hipGetErrorString(code) << ' ';
          std::cout << "in " << name << std::endl;
          assert(false);
      }
  };

  check(hipGetLastError());
}

void test_wrapper(){
  test<<<40,256>>>();
  assert_if_error("");
}