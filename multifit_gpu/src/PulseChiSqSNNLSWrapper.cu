#include "multifit_gpu/interface/PulseChiSqSNNLSWrapper.h"
#include "multifit_gpu/interface/EigenMatrixTypes.h"
#include "multifit_gpu/interface/PulseChiSqSNNLS.h"

// PulseChiSqSNNLSWrapper::PulseChiSqSNNLSWrapper(){};
// PulseChiSqSNNLSWrapper::~PulseChiSqSNNLSWrapper(){};

// void PulseChiSqSNNLSWrapper::disableErrorCalculation(){
//     pulseChiSq.disableErrorCalculation();
// }

// void PulseChiSqSNNLSWrapper::DoFit(DoFitArgs* args, bool* status){
//     GpuDoFit<<<1,1>>>(&pulseChiSq, args, status);
// }

DoFitResults* doFitWrapper(DoFitArgs* args, unsigned int n){
    DoFitArgs* deviceArgs;
    hipMalloc((void**) &deviceArgs, sizeof(DoFitArgs)*n);
    hipMemcpy(deviceArgs, args, sizeof(DoFitArgs)*n, hipMemcpyHostToDevice);
    DoFitResults* deviceResults;
    DoFitResults* results;
    hipMalloc((void**) &deviceResults, sizeof(DoFitResults)*n);
    GpuDoFit<<<1,1>>>(deviceArgs, deviceResults, n);
    hipMemcpy(deviceResults, results, sizeof(DoFitResults)*n, hipMemcpyDeviceToHost);
    hipFree(deviceArgs);
    hipFree(deviceResults);
    return results;
}
