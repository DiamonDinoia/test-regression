#include "multifit_gpu/interface/PulseChiSqSNNLSWrapper.h"

PulseChiSqSNNLSWrapper::PulseChiSqSNNLSWrapper(){};
PulseChiSqSNNLSWrapper::~PulseChiSqSNNLSWrapper(){};

void PulseChiSqSNNLSWrapper::disableErrorCalculation(){
    pulseChiSq.disableErrorCalculation();
}

void PulseChiSqSNNLSWrapper::DoFit(DoFitArgs* args, bool* status){
    GpuDoFit<<<1,1>>>(&pulseChiSq, args, status);
}
