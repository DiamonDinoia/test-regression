#include "multifit_gpu/interface/PulseChiSqSNNLSWrapper.h"
#include "multifit_gpu/interface/PulseChiSqSNNLS.h"

#include <vector>
#include <iostream>
#include <string>

std::vector<DoFitResults> doFitWrapper(std::vector<DoFitArgs> const& vargs) {
    // input parameters to the multifit on gpu
    DoFitArgs* d_args;
    DoFitResults* d_results;
    std::vector<DoFitResults> results;
    std::cout << "sizeof (DoFitResults) = " << sizeof(DoFitResults) << std::endl;
    std::cout << "sizeof (BXVector) = " << sizeof(BXVector) << std::endl;
    std::cout << "sizeof (PulseVector) = " << sizeof(PulseVector) << std::endl;
    std::cout << "vargs.size() = " << vargs.size() << std::endl;
    results.resize(100);
    std::cout << "size = " << results.size() << std::endl;
    std::cout << "capacity = " << results.capacity() << std::endl;

    // allocate on the device
    /*
    std::cout << "allocate on the device" << std::endl;
    hipMalloc((void**) &d_args, sizeof(DoFitArgs) * vargs.size());
    hipMalloc((void**) &d_results, sizeof(DoFitResults) * vargs.size());

    // transfer to the device
    std::cout << "copy to the device " << std::endl;
    hipMemcpy(d_args, vargs.data(), sizeof(DoFitArgs) * vargs.size(), hipMemcpyHostToDevice);

    // kernel invoacation
    std::cout << "launch the kenrel" << std::endl;
    int nthreadsPerBlock = 256;
    int nblocks = (vargs.size() + nthreadsPerBlock - 1) / nthreadsPerBlock;
    kernel_multifit<<<nblocks, nthreadsPerBlock>>>(d_args, d_results, vargs.size());
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    std::string name = "multifit_gpu";
    if (err != hipSuccess) {
        std::cout << "cuda eror!" << std::endl
            << hipGetErrorString(err) << std::endl;
        std::cout << "test " << name << " failed" << std::endl;
    }

    // copy results back
    std::cout << "copy back to the host" << std::endl;
    hipMemcpy(&(results[0]), d_results, sizeof(DoFitResults) * results.size(), hipMemcpyDeviceToHost);
    std::cout << "vresults.size() = " << results.size() << std::endl;

    // free resources
    std::cout << "free the device memory" << std::endl;
    hipFree(d_args);
    hipFree(d_results);
    */

    return std::vector<DoFitResults>{};
}
