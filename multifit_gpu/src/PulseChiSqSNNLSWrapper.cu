#include "multifit_gpu/interface/PulseChiSqSNNLSWrapper.h"

PulseChiSqSNNLSWrapper::PulseChiSqSNNLSWrapper(){};
PulseChiSqSNNLSWrapper::~PulseChiSqSNNLSWrapper(){};

void PulseChiSqSNNLSWrapper::disableErrorCalculation(){
    pulseChiSq.disableErrorCalculation();
}

void PulseChiSqSNNLSWrapper::DoFit(DoFitArgs* args, double* status){
    GpuDoFit<<<1,1>>>(&pulseChiSq, args, status);
}
