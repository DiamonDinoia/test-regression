#include "multifit_gpu/interface/PulseChiSqSNNLSWrapper.h"
#include "multifit_gpu/interface/PulseChiSqSNNLS.h"

#include <vector>
#include <iostream>

std::vector<DoFitResults> doFitWrapper(std::vector<DoFitArgs> const& vargs) {
    // input parameters to the multifit on gpu
    DoFitArgs* d_args;
    DoFitResults* d_results;
    std::vector<DoFitResults> results;

    // allocate on the device
    std::cout << "allocate on the device" << std::endl;
    hipMalloc((void**) &d_args, sizeof(DoFitArgs) * vargs.size());
    hipMalloc((void**) &d_results, sizeof(DoFitResults) * vargs.size());

    // transfer to the device
    std::cout << "copy to the device " << std::endl;
    hipMemcpy(d_args, vargs.data(), sizeof(DoFitArgs) * vargs.size(), hipMemcpyHostToDevice);

    // kernel invoacation
    std::cout << "launch the kenrel" << std::endl;;
    kernel_multifit<<<1,1>>>(d_args, d_results, vargs.size());
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "cuda eror!" << std::endl
            << hipGetErrorString(err) << std::endl;
        std::cout << "test " << name << " failed" << std::endl;
    }

    // copy results back
    std::cout << "copy back to the host" << std::endl;
    hipMemcpy(&results[0], d_args, sizeof(DoFitResults) * vargs.size(), hipMemcpyDeviceToHost);

    // free resources
    std::cout << "free the device memory" << std::endl;
    hipFree(d_args);
    hipFree(d_results);

    return results;
}
