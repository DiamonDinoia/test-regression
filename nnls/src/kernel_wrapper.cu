#include "../interface/kernel_wrapper.h"
#include "../interface/fnnls.h"
// #include "../interface/kernels.h"

#include <iostream>
#include <string>

using namespace std;

void assert_if_error(std::string const& name) {
  auto check = [&name](auto code) {
    if (code != hipSuccess) {
      std::cout << hipGetErrorString(code) << ' ';
      std::cout << "in " << name << std::endl;
      assert(false);
    }
  };

  check(hipGetLastError());
}

std::vector<FixedVector> fnnls_wrapper(std::vector<NNLS_args> const& args,
                                       double eps,
                                       unsigned int max_iterations) {
  // host solution vector
  std::vector<FixedVector> x(args.size());

  // device pointers
  NNLS_args* d_args;
  FixedVector* d_x;

  // arguments allocation
  hipMalloc((void**)&d_args, sizeof(NNLS_args) * args.size());
  // results allocation
  hipMalloc((void**)&d_x, sizeof(FixedVector) * args.size());

  // arguments copy
  hipMemcpy(d_args, args.data(), sizeof(NNLS_args) * args.size(),
             hipMemcpyHostToDevice);

  printf("launch kernel fnnls\n");
  int nthreadsPerBlock = 256;
  int nblocks = (args.size() + nthreadsPerBlock - 1) / nthreadsPerBlock;
  fnnls_kernel<<<nblocks, nthreadsPerBlock>>>(d_args, d_x, args.size(), eps,
                                              max_iterations);
  // fnnls_kernel<<<1,1>>>(d_args, d_x, args.size(), eps, max_iterations);
  hipDeviceSynchronize();
  assert_if_error("fnnls");
  printf("finish kernel fnnls\n");

  // copy the results back from the device
  hipMemcpy(x.data(), d_x, sizeof(FixedVector) * args.size(),
             hipMemcpyDeviceToHost);

  // clear and exit
  hipFree(d_args);
  hipFree(d_x);

  // for(const auto& result: x){
  // cout << "x" << endl;
  // cout << result.transpose() << endl;
  // break;
  // }

  return x;
}
