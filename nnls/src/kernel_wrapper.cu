#include "hip/hip_runtime.h"
#include "nnls/interface/kernel_wrapper.h"
#include "nnls/interface/fnnls.h"
#include "nnls/interface/inplace_fnnls.h"

#include <iostream>
#include <string>

using namespace std;

void assert_if_error(std::string const& name) {
  auto check = [&name](auto code) {
    if (code != hipSuccess) {
      std::cout << hipGetErrorString(code) << ' ';
      std::cout << "in " << name << std::endl;
      assert(false);
    }
  };

  check(hipGetLastError());
}

std::vector<FixedVector> fnnls_wrapper(std::vector<NNLS_args> const& args,
                                       double eps,
                                       unsigned int max_iterations) {
  // host solution vector
  std::vector<FixedVector> x(args.size());

  // device pointers
  NNLS_args* d_args;
  FixedVector* d_x;

  // arguments allocation
  hipMalloc((void**)&d_args, sizeof(NNLS_args) * args.size());
  // results allocation
  hipMalloc((void**)&d_x, sizeof(FixedVector) * args.size());

  // arguments copy
  hipMemcpy(d_args, args.data(), sizeof(NNLS_args) * args.size(),
             hipMemcpyHostToDevice);

  printf("launch kernel fnnls\n");
  int nthreadsPerBlock = 256;
  int nblocks = (args.size() + nthreadsPerBlock - 1) / nthreadsPerBlock;
  fnnls_kernel<<<nblocks, nthreadsPerBlock>>>(d_args, d_x, args.size(), eps,
                                              max_iterations);
  // fnnls_kernel<<<1,1>>>(d_args, d_x, args.size(), eps, max_iterations);
  hipDeviceSynchronize();
  assert_if_error("fnnls");
  printf("finish kernel fnnls\n");

  // copy the results back from the device
  hipMemcpy(x.data(), d_x, sizeof(FixedVector) * args.size(),
             hipMemcpyDeviceToHost);

  // clear and exit
  hipFree(d_args);
  hipFree(d_x);

  // for(const auto& result: x){
  // cout << "x" << endl;
  // cout << result.transpose() << endl;
  // break;
  // }

  return x;
}

__global__ void inplace_fnnls_kernel(NNLS_args* args,
                                   FixedVector* x,
                                      unsigned int n,
                                      double eps,
                              unsigned int max_iterations) {
   // thread idx
   // printf("hello nnls\n");
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   // printf("thread index %i n %i\n", i,n);
   if (i >= n)
         return;
    // printf("thread index %i n %i\n", i,n);
     
    auto& A = args[i].A;
    auto& b = args[i].b;
     
    // printf("inside the kernel\n");
    // print_fixed_matrix(A);
    // print_fixed_vector(b);
    inplace_fnnls(A, b, x[i], eps, max_iterations);
}

__global__ void fnnls_kernel(NNLS_args* args,
                                      FixedVector* x,
                                      unsigned int n,
                                      double eps,
                                      unsigned int max_iterations) {
    // thread idx
    // printf("hello nnls\n");
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   // printf("thread index %i n %i\n", i,n);
   if (i >= n)
         return;
    // printf("thread index %i n %i\n", i,n);
     
    auto& A = args[i].A;
    auto& b = args[i].b;
     
    // printf("inside the kernel\n");
    // print_fixed_matrix(A);
    // print_fixed_vector(b);
    fnnls(A, b, x[i], eps, max_iterations);
}
